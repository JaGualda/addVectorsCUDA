#include "hip/hip_runtime.h"
// Suma de vectores secuencial
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime_api.h>
#include <helper_timer.h>

StopWatchInterface *hTimer = NULL;
StopWatchInterface *kTimer = NULL;

typedef int *vector;



void LoadP(vector P, unsigned int n)
{
   unsigned int i;

   for (i=0;i<n;i++) 
     P[i] = i;
}

// Function for generating random values for a vector
void LoadStartValuesIntoVectorRand(vector V, unsigned int n)
{
   unsigned int i;

   for (i=0;i<n;i++) 
     V[i] = (int)(random()%9);
}


// Function for printing a vector
void PrintVector(vector V, unsigned int n)
{
   unsigned int i;

   for (i=0;i<n;i++)
      printf("%d\n",V[i]);
}

// Suma vectores cC = cA + cB
__global__ void SumVectorCuda(vector cA, vector cB, vector cC, vector cP, unsigned int n, unsigned int v)
{
   unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
   int end = ((idx + 1) * v) - 1;
   int tid /*= idx * v*/;
   //printf("End: %d\n", end);
   
   for (tid = idx * v; tid <= end ; tid++){
      //printf("Tid: %d, Thread: %d\n", tid, idx);
      //printf("Vector cA: %d Thread: %d\n", cA[tid], idx);
      cC[cP[tid]] = cA[cP[tid]] + cB[cP[tid]];
   }
}


// ------------------------
// MAIN function
// ------------------------
int main(int argc, char **argv)
{
   float timerValue;
   double ops;
   unsigned int n, v;

   //Pasar numero de componentes del vector por thread (v).
   if (argc == 3){
      n = atoi(argv[1]);
      v = atoi(argv[2]); 
   }  
   else
     {
       printf ("Sintaxis: <ejecutable> <total number of elements> <elementos del vector por thread>\n");
       exit(0);
     }

   if(n%v != 0){
      printf("El número de componentes del vector por thread no es divisor del total de elementos del mismo\n");
      exit(0);
   }

   srandom(12345);

   // Define vectors at host
   vector A;
   vector B;
   vector C;
   vector P;

   vector cA;
   vector cB;
   vector cC;
   vector cP;

   sdkCreateTimer(&hTimer);
   sdkResetTimer(&hTimer);
   sdkStartTimer(&hTimer);

   // Load values into A
   A = (int *) malloc(n*sizeof(int));
   hipMalloc((void**)&cA,n*sizeof(int));
   LoadStartValuesIntoVectorRand(A,n);
   hipMemcpy(cA, A, n*sizeof(int), hipMemcpyHostToDevice);
   //printf("\nPrinting Vector A  %d\n",n);
   //PrintVector(A,n);

   // Load values 
   B = (int *) malloc(n*sizeof(int));
   hipMalloc((void**)&cB,n*sizeof(int));
   LoadStartValuesIntoVectorRand(B,n);
   hipMemcpy(cB, B, n*sizeof(int), hipMemcpyHostToDevice);
   //printf("\nPrinting Vector B  %d\n",n);
   //PrintVector(B,n);

   C = (int *) malloc(n*sizeof(int));
   hipMalloc(&cC,n*sizeof(int));

    // Load values 
   P = (int *) malloc(n*sizeof(int));
   hipMalloc((void**)&cP,n*sizeof(int));
   LoadP(P,n);
   hipMemcpy(cP, P, n*sizeof(int), hipMemcpyHostToDevice);

   sdkCreateTimer(&kTimer);
   sdkResetTimer(&kTimer);
   sdkStartTimer(&kTimer);

   //printf("Llega\n");

   // execute the subprogram
   SumVectorCuda<<<n/(1024*v),1024>>>(cA,cB,cC,cP,n,v);

   hipDeviceSynchronize();

   sdkStopTimer(&kTimer);

   //printf("Llega 2\n");

   //Copiar de dispositivo a host
   hipMemcpy(C, cC, n*sizeof(int), hipMemcpyDeviceToHost);

   //printf("Copia\n");

   //printf("\nPrinting vector C  %d\n",n);
   //PrintVector(C,n);
   

   // Free vectors
   free(A);
   free(B);
   free(C);
   free(P);

   hipFree(cA);
   hipFree(cB);
   hipFree(cC);
   hipFree(cP);

   sdkStopTimer(&hTimer);

   //cambiar timers a los de cuda

   timerValue = sdkGetTimerValue(&kTimer);
   timerValue = timerValue / 1000;
   sdkDeleteTimer(&kTimer);
   printf("Tiempo kernel: %f s", timerValue);
   ops = n/timerValue;
   printf("    %f GFLOPS\n",(ops)/1000000000);
   timerValue = sdkGetTimerValue(&hTimer);
   timerValue = timerValue / 1000;
   sdkDeleteTimer(&hTimer);
   printf("Tiempo total: %f s", timerValue);
   ops = n/timerValue;
   printf("    %f GFLOPS \n",(ops)/1000000000);

   return 0;
}
